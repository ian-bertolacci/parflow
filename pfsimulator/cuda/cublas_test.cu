#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#define M 6
#define N 5
#define IDX2C(i,j,ld) (((j)*(ld))+(i))

static __inline__ void modify (hipblasHandle_t handle, float *m, int ldm, int n, int p, int q, float alpha, float beta){
    hipblasSscal (handle, n-q, &alpha, &m[IDX2C(p,q,ldm)], ldm);
    hipblasSscal (handle, ldm-p, &beta, &m[IDX2C(p,q,ldm)], 1);
}

extern "C"
void cublas_test() {
    printf("\nStart CUDA cublas_test!\n"); 

    hipError_t cudaStat;    
    hipblasStatus_t stat;
    hipblasHandle_t handle;
    int i, j;
    float* devPtrA;
    float* a = 0;
    // a = (float *)malloc (M * N * sizeof (*a));
    // if (!a) {
    //     printf ("host memory allocation failed");
    //     return;
    // }
    cudaStat = hipMallocManaged ((void**)&devPtrA, M*N*sizeof(float));
    if (cudaStat != hipSuccess) {
        printf ("device memory allocation failed");
        return;
    }
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            devPtrA[IDX2C(i,j,M)] = (float)(i * M + j + 1);
        }
    }
    stat = hipblasCreate(&handle);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("CUBLAS initialization failed\n");
        return;
    }
    // stat = cublasSetMatrix (M, N, sizeof(*a), a, M, devPtrA, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data download failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return;
    }
    modify (handle, devPtrA, M, N, 1, 2, 16.0f, 12.0f);
    // stat = cublasGetMatrix (M, N, sizeof(*a), devPtrA, M, a, M);
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        printf ("data upload failed");
        hipFree (devPtrA);
        hipblasDestroy(handle);
        return;
    }
    hipDeviceSynchronize();
    for (j = 0; j < N; j++) {
        for (i = 0; i < M; i++) {
            printf ("%7.0f", devPtrA[IDX2C(i,j,M)]);
        }
        printf ("\n");
    }
    hipFree (devPtrA);
    hipblasDestroy(handle);
    // free(a);
    printf("CUDA cublas_test finished!\n\n"); 
}